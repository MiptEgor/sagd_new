#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <thrust/extrema.h>
#define BLOCK_SIZE 16

float f(float g, float l)
{
	return (962361*(pow(-74500000*(-2 + g)*g + 220*l + 
     6*(31333357 - 3250013*g)*g*l + (-110 + 39000039*g)*l*l,2) / 2500 -
   1600*g*l*(1 + g + l)*(5811*g*g - 110*(-2 + l)*(149 + 39*l) - 
      2*g*(14006 + 1521*l))))/(100 * pow((1 + g + l),8));
}
__device__ float k(float tetta)
{
	return tetta * tetta;
}

__device__ float k_shtrih(float tetta)
{
	return tetta * tetta;
}

__device__ float W1(float l, float g, float rg, float rl, float xi) 
{
	return   l * (l - 2 - 2 * g) * (g * (rg - rl) - rl + 1) / pow(1 + g + l, 4);
}
__device__ float W2(float l, float g, float rg, float rl, float xi)
{
	return  - l * l * ((1 - 2 * g + l) * rg + (2 + 2  * g - l) * rl - 3) / pow(1 + g + l, 4);
}
__device__ float W3(float l, float g, float rg, float rl, float xi)
{
	return  g * g * ((g - 2 - 2 * l) * rg - (1 + g - 2 * l) * rl + 3) / pow(1 + g + l, 4) / xi;
}
__device__ float W4(float l, float g, float rg, float rl, float xi)
{
	return + g * (g - 2 * l - 2) * (rg + rg * l - l * rl - 1) / pow(1 + g + l, 4) / xi;
} 

__global__ void parallel_calculate(thrust::device_ptr<float> huge_dev, int N, float rg, float rl, float xi, float h)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	float g = i * h;
	float l = j * h;
	//float b = W1(l, g , rg, rl, xi) + W4(l, g , rg, rl, xi);
	if ((i < N)&&(j < N))
	{
		float D = powf( W1(l, g , rg, rl, xi) - W4(l, g , rg, rl, xi), 2) + 4 * W3(l, g , rg, rl, xi) * W2(l, g , rg, rl, xi) 
		 - 4 * W1(l, g , rg, rl, xi) * W4(l, g , rg, rl, xi);
	//float dd = powf(fabs(D), 0.5f);
		huge_dev[i + j * N] = D;
		if (D>0)
			huge_dev[i + j * N] = 0;
	/*if (D < dev[2])
	{
		dev[0] = g;
		dev[1] = l;
		dev[2] = D;
		dev[3] = b;
		dev[4] = dd;
	}*/
	}
	__syncthreads();

}


template<typename T>
void put_be(std::ofstream &f, const T val) {
    union {
        T value;
        char bytes[sizeof(T)];
    } x;
    x.value = val;
    std::reverse(x.bytes, x.bytes + sizeof(T));
    f.write(x.bytes, sizeof(T));
}

void print(int nx, int ny, thrust::host_vector<float>::iterator huge_host, float h)
	{

		int Nx = nx;
		int Ny = ny;
	    char path[1024];
	    sprintf(path, "res.vtk");
	    std::ofstream f(path, std::ios::binary);

	    f << "# vtk DataFile Version 3.0" << std::endl;
	    f << "Comment" << std::endl;
	    f << "BINARY" << std::endl;
	    f << "DATASET RECTILINEAR_GRID" << std::endl;
	    f << "DIMENSIONS " << Nx << " " << Ny << " 1" << std::endl;
	    f << "X_COORDINATES " << Nx << " float" << std::endl;
	    for (size_t i = 0; i < Nx; i++)
	        put_be<float>(f, h * i);
	    f << "Y_COORDINATES " << Ny << " float" << std::endl;
	    for (size_t j = 0; j < Ny; j++)
	        put_be<float>(f, h * j);
	    f << "Z_COORDINATES 1 float" << std::endl;
	    put_be<float>(f, 0);

	    f << "POINT_DATA " << Nx * Ny << std::endl;
	    f << "SCALARS p float\nLOOKUP_TABLE default" << std::endl;
	    for (size_t j = 0; j < Ny; j++)
	        for (size_t i = 0; i < Nx; i++)
	            put_be<float>(f, huge_host[i + j * Nx]);
	    f.close();

	}

int main(int argc, char const *argv[])
{
	std::cout.precision(10);
	int N = 2e3;
	float h = 1.e2 / N;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 nblocks((int)(N) / BLOCK_SIZE + 1, (int)(N) / BLOCK_SIZE + 1);
	thrust::device_vector<float> dev(5, 0.f);
	thrust::host_vector<float> host(5, 0.f);
	thrust::device_vector<float> huge_dev(N * N, 0.f);
	thrust::host_vector<float> huge_host(N * N, 0.f);
	float rl = 0.3;
	float rg = 0.02;
	float xi = 1.e-4;
	std::cout<<"g"<<" "<<"l"<<" "<<"D"<<" "<<"b"<<" "<<"dd" <<std::endl;
	parallel_calculate<<<nblocks, block>>>(huge_dev.data(), N, rg, rl, xi, h);
	thrust::copy(huge_dev.begin(), huge_dev.end(), huge_host.begin());
	//std::cout<<huge_host[0]<<" "<<huge_host[1]<<" "<<huge_host[2]<<" "<<huge_host[3]<<" "<<huge_host[4] <<std::endl;
	print(N, N, huge_host.data(), h);
	float min = *(thrust::min_element(huge_dev.begin(), huge_dev.end()));
	std::cout<<min<<std::endl;
	/*xi = 1e-2;
	parallel_calculate<<<nblocks, block>>>(dev.data(), N, rg, rl, xi, h);
	thrust::copy(dev.begin(), dev.end(), host.begin());
	std::cout<<host[0]<<" "<<host[1]<<" "<<host[2]<<" "<<host[3]<<" "<<host[4] <<std::endl;
	*/
	std::cout<<"end"<<std::endl;
	return 0;
}